
#include <hip/hip_runtime.h>
__global__ void kernel_A(double *A, int N, int M)
{
    double d = 0.0;
    double e = 0.0;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N)
    {

#pragma unroll(100)
        for (int j = 0; j < M; ++j)
        {
            d += A[idx];
            e += A[idx];
        }

        A[idx] = d + e;
    }
}

__global__ void kernel_B(double *A, int N, int M)
{
    double d = 0.0;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N)
    {

#pragma unroll(100)
        for (int j = 0; j < M; ++j)
        {
            d += A[idx];
        }

        A[idx] = d;
    }
}

int main()
{

    double *A;

    int N = 80 * 2048 * 100; // 100 * maximum number of resident threads on V100
    size_t sz = N * sizeof(double);

    hipMalloc((void **)&A, sz);

    hipMemset(A, 1, sz);

    int threadsPerBlock = 64;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    int M = 1000;

    kernel_A<<<numBlocks, threadsPerBlock>>>(A, N, M);
    kernel_B<<<numBlocks, threadsPerBlock>>>(A, N, M);

    hipDeviceSynchronize();
}
